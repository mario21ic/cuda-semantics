#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NELEMENTS 12
// NRUNS should be < NELEMENTS.
#define NRUNS 2
#define NBLOCKS 2
// NTHREADS_PER_BLOCK*NBLOCKS should equal NELEMENTS
#define NTHREADS_PER_BLOCK 6

__global__ void sum_kernel(int* g_odata, int* g_idata, int run) {
      __shared__ int shared[NELEMENTS];
      int i, gtid = blockIdx.x * blockDim.x + threadIdx.x;
      int tid = threadIdx.x;

      shared[tid] = g_idata[gtid];
      
      __syncthreads();

      if (tid < NTHREADS_PER_BLOCK/2) {
            shared[tid] += shared[NTHREADS_PER_BLOCK/2 + tid];
      }

      __syncthreads();

      if (tid == 0) {
            for (i = 1; i != NTHREADS_PER_BLOCK/2; ++i) {
                  shared[0] += shared[i];
            }

            if (gtid == 0) {
                  g_odata[run] = shared[0];
            } else {
                  __threadfence();
                  g_odata[run] += shared[0];
            }
      }
}

int main(int argc, char** argv) {
      int* d_idata, *d_odata, *h_data;
      int i;
      dim3 grid;
      dim3 block;

      // Use a different stream for every run.
      hipStream_t streams[NRUNS];

      grid.x = NBLOCKS;
      grid.y = 1;
      grid.z = 1;

      block.x = NTHREADS_PER_BLOCK;
      block.y = 1;
      block.z = 1;

      h_data = (int*)malloc(NELEMENTS * sizeof(int));

      printf("INPUT: ");
      for(i = 0; i != NELEMENTS; ++i) {

            h_data[i] = (11 + i * i) % 7;
            printf(" %d ", h_data[i]);
      }
      printf("\n");

      hipMalloc(&d_idata, NELEMENTS * sizeof(int));
      hipMalloc(&d_odata, NRUNS * sizeof(int));

      hipMemcpy(d_idata, h_data, NELEMENTS * sizeof(int), hipMemcpyHostToDevice);

      hipMemset(d_odata, 0, NRUNS * sizeof(int));
      
      printf("Launching %d blocks of %d threads each " 
             "to asychronously sum the list above %d times.\n", 
             NBLOCKS, NTHREADS_PER_BLOCK, NRUNS);

      hipDeviceSynchronize();
      for (i = 0; i != NRUNS; ++i) {
            hipStreamCreate(&streams[i]);
            sum_kernel<<< grid, block, NELEMENTS * sizeof(int), streams[i] >>>
                  (d_odata, d_idata, i);
      }
      hipDeviceSynchronize();

      hipMemcpyAsync(h_data, d_odata, NRUNS * sizeof(int), hipMemcpyDeviceToHost, streams[0]);

      hipStreamSynchronize(streams[0]);
      hipDeviceSynchronize();

      printf("OUTPUT: ");
      for(i = 0; i != NRUNS; ++i) {
            hipStreamDestroy(streams[i]);
            printf(" %d ", h_data[i]);
      }
      printf("\n");

      free(h_data);
      hipFree(d_idata);
      hipFree(d_odata);
}